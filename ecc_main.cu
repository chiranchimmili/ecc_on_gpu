#include <iostream>
#include <hip/hip_runtime.h>
#include "ecc.cuh"
#include "bfs_new.cuh"

void checkCudaError(hipError_t error, const char *message) {
    if (error != hipSuccess) {
        std::cerr << message << ": " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char** argv) {

    BFSGraph(argc, argv);

    uint8_t *h_data, *d_data, *d_encoded_data, *d_corrected_data;

    size_t total_rows = TOTAL_ROWS; // 1GB / 256 bytes per row

    h_data = new uint8_t[DATA_LEN * total_rows];

    hipMalloc((void **)&d_data, DATA_LEN * total_rows);
    hipMalloc((void **)&d_encoded_data, TOTAL_LEN * total_rows);
    hipMalloc((void **)&d_corrected_data, DATA_LEN * total_rows);

    generate_data(h_data, total_rows);

    hipMemcpy(d_data, h_data, DATA_LEN * total_rows, hipMemcpyHostToDevice);

    hipStream_t stream1, stream2;
    int leastPriority, greatestPriority;
    hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);

    hipStreamCreateWithPriority(&stream1, hipStreamDefault, greatestPriority);
    hipStreamCreateWithPriority(&stream2, hipStreamDefault, leastPriority);

    int threads_per_block = 256;
    size_t num_blocks = (total_rows + threads_per_block - 1) / threads_per_block;
    rs_encode<<<num_blocks, threads_per_block, 0, stream2>>>(d_data, d_encoded_data, total_rows);

    bool stop = false;
    do {
        stop = false;
        hipMemcpyAsync(d_over, &stop, sizeof(bool), hipMemcpyHostToDevice, stream1);

        Kernel<<<num_of_blocks, MAX_THREADS_PER_BLOCK, 0, stream1>>>(d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
        Kernel2<<<num_of_blocks, MAX_THREADS_PER_BLOCK, 0, stream1>>>(d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);

        hipMemcpyAsync(&stop, d_over, sizeof(bool), hipMemcpyDeviceToHost, stream1);
        hipStreamSynchronize(stream1);
    } while(stop);


    rs_decode<<<num_blocks, threads_per_block, 0, stream2>>>(d_encoded_data, d_corrected_data, total_rows);

    hipStreamSynchronize(stream2);
    hipStreamSynchronize(stream1);


    checkCudaError(hipFree(d_data), "Failed to free device memory for data");
    checkCudaError(hipFree(d_encoded_data), "Failed to free device memory for encoded data");
    checkCudaError(hipFree(d_corrected_data), "Failed to free device memory for corrected data");

    delete[] h_data;

    checkCudaError(hipStreamDestroy(stream1), "Failed to destroy stream1");
    checkCudaError(hipStreamDestroy(stream2), "Failed to destroy stream2");

    return 0;
}
